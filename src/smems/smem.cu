#include "hip/hip_runtime.h"
#include "smem.h"
#include "FMI_search.h"

SMEM *SMEMSerach::collect_smem(const bseq1_t *seq, int nseq, int32_t min_interval)
{
    /* init  SMEMS status*/
    int current_seq_id = 0;
    int min_seq_id = std::min(batch_smems_size, nseq);
    int running_reads = 0;
    int max_length = -1;

    for (int i = 0; i < nseq; i++) {
        if (seq[i].l_seq > max_length) {
            max_length = seq[i].l_seq;
        }
    }
    if (max_length * nseq > max_pre_num) {
        prev = (SMEM *)realloc(prev, max_length * sizeof(SMEM));
    }
    // TODO: resource reside in object
    int *running_idx = (int *)malloc(sizeof(int) * batch_smems_size);
    int *idle_idx = (int *)malloc(sizeof(int) * batch_smems_size);
    int *prev_num_ = (int *)malloc(sizeof(int) * batch_smems_size);

    while (current_seq_id < min_seq_id) {
        int next_i = 0;
        while (seq[current_seq_id].seq[next_i] >= 4 || seq[current_seq_id].seq[next_i + 1] >= 4) next_i++;
        uint8_t c = seq[current_seq_id].seq[next_i];
        SMEM *tmp = prev + (current_seq_id * max_length);
        status[current_seq_id].rid = current_seq_id;
        status[current_seq_id].step_status = StepStatus::first_pass;
        status[current_seq_id].direct_status = SearchDirectionStatus::foward;
        status[current_seq_id].seq_offset = 1;
        host_bases[current_seq_id] = 3 - seq[current_seq_id].seq[1];
        host_smems[current_seq_id].k = count[3 - c];
        host_smems[current_seq_id].l = count[c];
        host_smems[current_seq_id].s = count[c + 1] - count[c];
        tmp->m = tmp->n = 0;
        tmp->rid = current_seq_id;
        tmp->s = host_smems[current_seq_id].s;
        prev_num_[current_seq_id] = 0;
        current_seq_id++;
        running_reads++;
    }
    /* prepare smems array */
    int new_idle_idx = 0;
    do {
        // TODO: less  use cpu method
        backward(running_reads);
        int new_running_idx = 0;
        for (int i = 0; i < running_reads; i++) {
            SMEMS_STATUS *curr = status + running_idx[i];
            // update host MEMS  and Modify idle_idx
            if (curr->step_status != StepStatus::bwt_seed_strategy) {
                if (curr->direct_status == SearchDirectionStatus::foward) {
                    // todo: l_seq
                    int mapping_prev_offset = i * max_length;
                    //  before backward may save
                    prev_num_[i] += host_smems[i].s != prev[mapping_prev_offset + prev_num_[i]].s;
                    SMEM *current_pre = prev + mapping_prev_offset + prev_num_[i];
                    current_pre->s = host_smems[i].s;
                    current_pre->k = host_smems[i].l;
                    current_pre->l = host_smems[i].k;
                    current_pre->n = curr->seq_offset;
                    current_pre->m = curr->anchor;
                    prev[mapping_prev_offset + prev_num_[i]].n = curr->seq_offset;
                    if (host_smems[i].s < min_interval || curr->seq_offset == seq[curr->rid].l_seq - 1) {  // switch to BackWard
                        // Last interval
                        if (curr->seq_offset != seq[curr->rid].l_seq - 1)
                            prev_num_[i] += (prev[mapping_prev_offset + prev_num_[i]].s >= min_interval);
                        // SORT ...
                        for (int p = 0; p < (prev_num_[i] / 2); p++) {
                            SMEM temp = prev[mapping_prev_offset + p];
                            prev[mapping_prev_offset + p] = prev[mapping_prev_offset + prev_num_[i] - p - 1];
                            prev[mapping_prev_offset + prev_num_[i] - p - 1] = temp;
                        }
                        curr->rightmost = curr->seq_offset;
                        if (curr->anchor == 0 || prev_num_[i] == 0) {  // continue  foward search from next anchor's
                            while (seq[curr->rid].seq[curr->seq_offset] >= 4 || seq[curr->rid].seq[curr->seq_offset + 1] >= 4)
                                curr->seq_offset++;
                            curr->anchor = curr->seq_offset + 1;
                            uint8_t c = seq[curr->rid].seq[curr->anchor];
                            host_bases[new_running_idx] = 3 - seq[curr->rid].seq[curr->anchor + 1];
                            host_smems[new_running_idx].k = count[3 - c];
                            host_smems[new_running_idx].l = count[c];
                            host_smems[new_running_idx].s = count[c + 1] - count[c];
                            curr->seq_offset = curr->anchor + 1;
                            running_idx[new_running_idx] = running_idx[i];
                            new_running_idx++;
                            curr->direct_status = SearchDirectionStatus::foward;
                        }
                        else {
                            // starting backward search
                            curr->seq_offset = curr->anchor - 1;
                            host_bases[new_running_idx] = seq[curr->rid].seq[curr->seq_offset];
                            running_idx[new_running_idx] = running_idx[i];
                            host_smems[new_running_idx].k = prev[mapping_prev_offset].k;
                            host_smems[new_running_idx].l = prev[mapping_prev_offset].l;
                            host_smems[new_running_idx].s = prev[mapping_prev_offset].s;
                            new_running_idx++;
                            curr->currr_offset = 0;
                            curr->prev_offset = 0;
                            curr->direct_status = SearchDirectionStatus::backward;
                        }
                    }
                    else {
                        //  continue Foward.
                        curr->seq_offset++;
                        host_bases[new_running_idx] = 3 - seq[curr->rid].seq[curr->seq_offset];
                        host_smems[new_running_idx].k = host_smems[mapping_prev_offset].l;
                        host_smems[new_running_idx].l = host_smems[mapping_prev_offset].k;
                        host_smems[new_running_idx].s = host_smems[mapping_prev_offset].s;
                        running_idx[new_running_idx] = running_idx[i];
                        new_running_idx++;
                    }
                }
                else {
                    if (curr->prev_offset == prev_num_[i] - 1) {
                        if (curr->currr_offset == 0) {  // backward can't extend , switch new step
                            if (curr->rightmost == seq[curr->rid].l_seq - 1) {
                                if (curr->step_status == StepStatus::first_pass) {
                                    idle_idx[new_idle_idx] = running_idx[i];
                                    new_idle_idx++;
                                }
                                else {
                                    // swith bwt seeds
                                    curr->step_status = StepStatus::bwt_seed_strategy;
                                    int next_i = 0;
                                    while (seq[curr->rid].seq[next_i] >= 4 || seq[curr->rid].seq[next_i + 1] >= 4) next_i++;
                                    uint8_t c = seq[curr->rid].seq[next_i + 1];
                                    host_bases[new_running_idx] = 3 - seq[curr->rid].seq[next_i];
                                    host_smems[new_running_idx].k = count[c];
                                    host_smems[new_running_idx].l = count[3 - c];
                                    host_smems[new_running_idx].s = count[c + 1] - count[c];
                                    curr->seq_offset = next_i + 1;
                                    new_running_idx++;
                                }
                            }
                            else {  // Switch forward
                                curr->anchor = curr->rightmost;
                                host_bases[new_running_idx] = 3 - seq[curr->rid].seq[curr->anchor + 1];
                                host_smems[new_running_idx].k = count[3 - seq[curr->rid].seq[curr->anchor]];
                                host_smems[new_running_idx].l = count[seq[curr->rid].seq[curr->anchor]];
                                host_smems[new_running_idx].s =
                                    count[seq[curr->rid].seq[curr->anchor] + 1] - count[seq[curr->rid].seq[curr->anchor]];
                                curr->seq_offset = curr->rightmost + 1;
                                running_idx[new_running_idx] = running_idx[i];
                                new_running_idx++;
                            }
                        }
                        else {  // continue backward
                            curr->seq_offset--;
                            host_bases[new_running_idx] = seq[curr->rid].seq[curr->seq_offset];
                            host_smems[new_running_idx].k = prev[i * max_length].k;
                            host_smems[new_running_idx].l = prev[i * max_length].l;
                            host_smems[new_running_idx].s = prev[i * max_length].s;
                            prev_num_[i] = curr->currr_offset;
                            curr->currr_offset = 0;
                            curr->prev_offset = 0;
                            running_idx[new_running_idx] = running_idx[i];
                            new_running_idx++;
                        }
                    }
                    else {
                        if (prev[i * max_length + curr->prev_offset].s != host_smems[i].s) {
                            if (!curr->has_optimal_smems_occurred && host_smems[i].s < min_interval &&

                                curr->rightmost - curr->seq_offset > 0) {
                                // push to results
                                curr->has_optimal_smems_occurred = true;
                                result[result_num].k = host_smems[i].k;
                                result[result_num].l = host_smems[i].l;
                                result[result_num].s = host_smems[i].s;
                                result[result_num].rid = curr->rid;
                                result[result_num].m = curr->seq_offset;
                                result[result_num].n = prev[i * max_length + curr->prev_offset].n;
                                // push rightmost
                                first_result[first_result_num].rid = curr->rid;
                                first_result[first_result_num].rightmost = curr->rightmost;
                                first_result_num++;
                                result_num++;
                            }

                            if (host_smems[i].s >= min_interval) {
                                prev[i * max_length + curr->currr_offset].k = host_smems[i].k;
                                prev[i * max_length + curr->currr_offset].l = host_smems[i].l;
                                prev[i * max_length + curr->currr_offset].s = host_smems[i].s;
                                curr->currr_offset++;
                            }
                        }
                        curr->prev_offset++;
                        host_bases[new_running_idx] = seq[curr->rid].seq[curr->seq_offset];
                        host_smems[new_running_idx].k = prev[i * max_length + curr->prev_offset].k;
                        host_smems[new_running_idx].l = prev[i * max_length + curr->prev_offset].l;
                        host_smems[new_running_idx].s = prev[i * max_length + curr->prev_offset].s;
                        running_idx[new_running_idx] = running_idx[i];
                        new_running_idx++;
                    }
                }
            }
            else {
                curr->seq_offset++;
                if (host_smems[new_running_idx].s < min_interval) {
                    if (prev[i * max_length].s > 0) {  // store
                        result[result_num].k = prev[i * max_length].k;
                        result[result_num].l = prev[i * max_length].l;
                        result[result_num].s = prev[i * max_length].s;
                        result[result_num].rid = curr->rid;
                        result[result_num].m = curr->seq_offset - 1;
                        result[result_num].n = curr->anchor;
                    }
                    curr->anchor = curr->seq_offset + 1;
                    host_bases[new_running_idx] = 3 - seq[curr->rid].seq[curr->anchor + 1];
                    host_smems[new_running_idx].k = count[3 - seq[curr->rid].seq[curr->anchor]];
                    host_smems[new_running_idx].l = count[seq[curr->rid].seq[curr->anchor]];
                    host_smems[new_running_idx].s = count[seq[curr->rid].seq[curr->anchor] + 1] - count[seq[curr->rid].seq[curr->anchor]];
                    curr->seq_offset = curr->anchor + 1;
                    running_idx[new_running_idx] = running_idx[i];
                    new_running_idx++;
                }
                else {
                    host_bases[new_running_idx] = 3 - seq[curr->rid].seq[curr->seq_offset];
                    // FIXME: swap  i == new_running_idx
                    host_smems[new_running_idx].k = host_smems[i].l;
                    host_smems[new_running_idx].l = host_smems[i].k;
                    host_smems[new_running_idx].s = host_smems[i].s;
                    prev[i * max_length].k = host_smems[new_running_idx].k;
                    prev[i * max_length].l = host_smems[new_running_idx].l;
                    prev[i * max_length].s = host_smems[new_running_idx].s;
                    running_idx[new_running_idx] = running_idx[i];
                }
                new_running_idx++;
            }
        }
        //  prepare new reads's array
        while (new_running_idx < batch_smems_size && (current_seq_id < nseq || first_result_num > 0)) {
            int new_idx = idle_idx[new_idle_idx];
            if (first_result_num > 0) {
                int rid = first_result[first_result_num].rid;
                int anchor = first_result[first_result_num].rightmost;
                status[new_idx].rid = rid;
                status[new_idx].step_status = StepStatus::second_pass;
                status[new_idx].direct_status = SearchDirectionStatus::foward;
                status[new_idx].seq_offset = anchor + 1;
                host_bases[new_idx] = 3 - seq[rid].seq[anchor + 1];
                host_smems[new_idx].k = count[seq[rid].seq[anchor]];
                host_smems[new_idx].l = count[3 - seq[rid].seq[anchor]];
                host_smems[new_idx].s = count[seq[rid].seq[anchor] + 1] - count[seq[rid].seq[anchor]];
                first_result_num--;
            }
            else {
                status[new_idx].rid = current_seq_id;
                status[new_idx].step_status = StepStatus::first_pass;
                status[new_idx].direct_status = SearchDirectionStatus::foward;
                status[new_idx].seq_offset = 1;
                host_bases[new_idx] = 3 - seq[current_seq_id].seq[1];
                host_smems[new_idx].k = count[seq[current_seq_id].seq[0]];
                host_smems[new_idx].l = count[3 - seq[current_seq_id].seq[0]];
                host_smems[new_idx].s = count[seq[current_seq_id].seq[0] + 1] - count[seq[current_seq_id].seq[0]];
                current_seq_id++;
            }
            running_idx[new_running_idx] = new_idx;
            new_running_idx++;
            new_idle_idx--;
        }
        running_reads = new_running_idx;
    } while (running_reads > 0);
    free(running_idx);
    free(idle_idx);
    free(prev_num_);
    return NULL;
}
/* TODO: Stream*/
void SMEMSerach::backward(int process_number)
{
    if (process_number == 0) return;
    hipMemcpy(device_bases, host_bases, process_number * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpy(device_smems, host_smems, process_number * sizeof(SMEM_CUDA), hipMemcpyHostToDevice);
    dim3 block(thread_per_block);
    dim3 grid(block_number);
    getOCC4Back<<<grid, block>>>(cp_occ, device_smems, bwt_mask_device, device_bases, process_number, sentinel_index);
    hipMemcpy(host_smems, device_smems, process_number * sizeof(SMEM_CUDA), hipMemcpyDeviceToHost);

    for (int i = 0; i < process_number; i++) {
        host_smems[i].k += count[host_bases[i]];
    }
}
#define IS_K(tid)                (tid < 15)
#define GET_GROUP_THREAD_ID(tid) (tid & 0x3)
#define GET_BASE_PAIR(tid)       ((tid >> 2) & 0x3)

/**
 * @brief backwardExt GPU implement
 * @param cp_occ  checkpoint occ scalar
 * @param bwt_mask  bwt mask array
 * @param bases  multipy base pairs 2bit encode
 * @param size  number of base pairs
 * @param sentinel_index  sentinel index in suffix array
 * @return void
 */
__global__ void getOCC4Back(CP_OCC *cp_occ, SMEM_CUDA *smems, unsigned short *bwt_mask, uint8_t *bases, int size, int64_t sentinel_index)
{
    int base_idx = blockIdx.x;
    int tid = threadIdx.x;
    unsigned short mask = 0;
    unsigned short onehot = 0;
    uint8_t count = 0;
    __shared__ int64_t k[4], l[4], s[4];
    if (base_idx >= size) {
        return;
    }
    uint8_t base = bases[base_idx];
    SMEM_CUDA curr = smems[base_idx];
    if (IS_K(tid)) {
        mask = bwt_mask[((curr.k & CP_MASK) << 2) + GET_GROUP_THREAD_ID(tid)];
        onehot = cp_occ[curr.k >> CP_SHIFT].one_hot_bwt_str[GET_BASE_PAIR(tid)] >> ((3 - GET_GROUP_THREAD_ID(tid)) << 4);
    }

    else {
        mask = bwt_mask[(((curr.k + curr.s) & CP_MASK) << 2) + GET_GROUP_THREAD_ID(tid)];
        onehot = cp_occ[(curr.k + curr.s) >> CP_SHIFT].one_hot_bwt_str[GET_BASE_PAIR(tid)] >> ((3 - GET_GROUP_THREAD_ID(tid)) << 4);
    }

    onehot = onehot & mask;
    /*TODO: bit opt */
    for (int i = 0; i < 16; i++) {
        count += (onehot & 0x1);
        onehot = onehot >> 1;
    }
    unsigned int wrap_mask = (0xFu) << ((tid >> 2) << 2);  //  Generate Wrap Mask for sync add count

    count += __shfl_xor_sync(wrap_mask, count, 1);
    count += __shfl_xor_sync(wrap_mask, count, 2);
    if ((GET_GROUP_THREAD_ID(tid)) == 0) {
        if (IS_K(tid)) {  // update k
            k[GET_BASE_PAIR(tid)] = count + cp_occ[curr.k >> CP_SHIFT].cp_count[GET_BASE_PAIR(tid)];
        }
        else {  // update L
            l[GET_BASE_PAIR(tid)] = count + cp_occ[(curr.k + curr.s) >> CP_SHIFT].cp_count[GET_BASE_PAIR(tid)];
        }
    }
    __syncthreads();
    if (tid < 4) {
        s[tid] = l[tid] - k[tid];
    }
    __syncthreads();
    if (tid == 0) {
        l[3] = curr.l + ((curr.k <= sentinel_index) && ((curr.k + curr.s) > sentinel_index));
        l[2] = l[3] + s[3];
        l[1] = l[2] + s[2];
        l[0] = l[1] + s[1];
        smems[base_idx].k = k[base];  // Need add COUNT
        smems[base_idx].l = l[base];
        smems[base_idx].s = s[base];
    }
    __syncthreads();
}
__device__ __host__ uint8_t countSetBits_loop(unsigned short n)
{
    uint8_t count = 0;
    for (int i = 0; i < 16; i++) {
        count += (n & 0x1);
        n = n >> 1;
    }
    return count;
}

__device__ __host__ uint8_t countSetBits_v1(unsigned short n)
{
    n = (n & 0x5555) + ((n >> 1) & 0x5555);
    n = (n & 0x3333) + ((n >> 2) & 0x3333);
    n = (n & 0x0f0f) + ((n >> 4) & 0x0f0f);
    n = (n & 0x00ff) + ((n >> 8) & 0x00ff);
    return n & 0x001f;
}

__device__ __host__ uint8_t countSetBits_v2(unsigned short n)
{
    n = (n & 0x5555) + ((n >> 1) & 0x5555);  // 2bit * 8
    n = (n & 0x3333) + ((n >> 2) & 0x3333);  // 4bit * 4
    n = (n & 0x0f0f) + ((n >> 4) & 0x0f0f);  // 8bit * 2
    return (n * 0x0101) >> 8;
}
