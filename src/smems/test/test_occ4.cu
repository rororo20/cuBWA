#include "FMI_search.h"
#include "gtest/gtest.h"

void generate_one_hot_mask(unsigned short bwt_mask[][4])
{
    uint64_t *one_hot_mask_array = (uint64_t *)_mm_malloc(64 * sizeof(uint64_t), 64);
    one_hot_mask_array[0] = 0;
    uint64_t base = 0x8000000000000000L;
    one_hot_mask_array[1] = base;
    int64_t i = 0;
    for (i = 2; i < 64; i++) {
        one_hot_mask_array[i] = (one_hot_mask_array[i - 1] >> 1) | base;
    }
    for (i = 0; i < 64; i++) {
        uint64_t offset = one_hot_mask_array[i];
        bwt_mask[i][0] = bwt_mask[i][1] = bwt_mask[i][2] = bwt_mask[i][3] = 0;
        for (int j = 0; j < 16; j++) {
            bwt_mask[i][0] = (bwt_mask[i][0] << 1) | (offset >> 63 & 0X1L);
            bwt_mask[i][1] = (bwt_mask[i][1] << 1) | (offset >> 62 & 0x1L);
            bwt_mask[i][2] = (bwt_mask[i][2] << 1) | (offset >> 61 & 0x1L);
            bwt_mask[i][3] = (bwt_mask[i][3] << 1) | (offset >> 60 & 0x1L);
            offset = offset << 4;
        }
    }
}

void generate_occ_cpp(CP_OCC &cpo, char base[64], int64_t cp_count[4])
{
    uint8_t enc_bases[64];
    cpo.cp_count[0] = cp_count[0];
    cpo.cp_count[1] = cp_count[1];
    cpo.cp_count[2] = cp_count[2];
    cpo.cp_count[3] = cp_count[3];

    for (int i = 0; i < 64; i++) {
        switch (base[i]) {
            case 'A':
                /* code */
                enc_bases[i] = 0;
                break;
            case 'C':
                enc_bases[i] = 1;
                break;
            case 'G':
                enc_bases[i] = 2;
                break;
            case 'T':
                enc_bases[i] = 3;
                break;
            default:
                enc_bases[i] = 4;
                break;
        }
    }
    cpo.one_hot_bwt_str[0] = 0;
    cpo.one_hot_bwt_str[1] = 0;
    cpo.one_hot_bwt_str[2] = 0;
    cpo.one_hot_bwt_str[3] = 0;
    for (int i = 0; i < 64; i++) {
        cpo.one_hot_bwt_str[0] = cpo.one_hot_bwt_str[0] << 1;
        cpo.one_hot_bwt_str[1] = cpo.one_hot_bwt_str[1] << 1;
        cpo.one_hot_bwt_str[2] = cpo.one_hot_bwt_str[2] << 1;
        cpo.one_hot_bwt_str[3] = cpo.one_hot_bwt_str[3] << 1;
        uint8_t c = enc_bases[i];
        if (c < 4) {
            cpo.one_hot_bwt_str[c] += 1;
        }
    }
}
class BackwardTest : public ::testing::Test
{
protected:
    static void SetUpTestSuite()
    {
        generate_one_hot_mask(bwt_mask);
        size_t array_size = sizeof(bwt_mask);
        hipMalloc(&bwt_mask_device, array_size);
        hipMemcpy(bwt_mask_device, bwt_mask, array_size, hipMemcpyHostToDevice);
    }
    static void TearDownTestSuite()
    {
        hipFree(bwt_mask_device);
    }
    void SetUp() {}
    void TearDown() {}
    static unsigned short bwt_mask[64][4];
    static unsigned short *bwt_mask_device;
    static CP_OCC *cpos;
    static CP_OCC *cpos_device;
    static int cpo_size;
};

unsigned short BackwardTest::bwt_mask[64][4];
unsigned short *BackwardTest::bwt_mask_device = NULL;
CP_OCC *BackwardTest::cpos = NULL;
CP_OCC *BackwardTest::cpos_device = NULL;

int BackwardTest::cpo_size = 1024 * 256;

TEST_F(BackwardTest, testcase1)
{
    EXPECT_EQ(bwt_mask[0][0], 0X0000);
    EXPECT_EQ(bwt_mask[0][1], 0X0000);
    EXPECT_EQ(bwt_mask[0][2], 0X0000);
    EXPECT_EQ(bwt_mask[0][3], 0X0000);

    EXPECT_EQ(bwt_mask[1][0], 0X8000);
    EXPECT_EQ(bwt_mask[1][1], 0X0000);
    EXPECT_EQ(bwt_mask[1][2], 0X0000);
    EXPECT_EQ(bwt_mask[1][3], 0X0000);

    EXPECT_EQ(bwt_mask[2][0], 0X8000);
    EXPECT_EQ(bwt_mask[2][1], 0X8000);
    EXPECT_EQ(bwt_mask[2][2], 0X0000);
    EXPECT_EQ(bwt_mask[2][3], 0X0000);

    EXPECT_EQ(bwt_mask[3][0], 0X8000);
    EXPECT_EQ(bwt_mask[3][1], 0X8000);
    EXPECT_EQ(bwt_mask[3][2], 0X8000);
    EXPECT_EQ(bwt_mask[3][3], 0X0000);

    EXPECT_EQ(bwt_mask[4][0], 0X8000);
    EXPECT_EQ(bwt_mask[4][1], 0X8000);
    EXPECT_EQ(bwt_mask[4][2], 0X8000);
    EXPECT_EQ(bwt_mask[4][3], 0X8000);

    EXPECT_EQ(bwt_mask[5][0], 0XC000);
    EXPECT_EQ(bwt_mask[5][1], 0X8000);
    EXPECT_EQ(bwt_mask[5][2], 0X8000);
    EXPECT_EQ(bwt_mask[5][3], 0X8000);

    EXPECT_EQ(bwt_mask[6][0], 0XC000);
    EXPECT_EQ(bwt_mask[6][1], 0XC000);
    EXPECT_EQ(bwt_mask[6][2], 0X8000);
    EXPECT_EQ(bwt_mask[6][3], 0X8000);

    EXPECT_EQ(bwt_mask[7][0], 0XC000);
    EXPECT_EQ(bwt_mask[7][1], 0XC000);
    EXPECT_EQ(bwt_mask[7][2], 0XC000);
    EXPECT_EQ(bwt_mask[7][3], 0X8000);

    EXPECT_EQ(bwt_mask[8][0], 0XC000);
    EXPECT_EQ(bwt_mask[8][1], 0XC000);
    EXPECT_EQ(bwt_mask[8][2], 0XC000);
    EXPECT_EQ(bwt_mask[8][3], 0XC000);

    EXPECT_EQ(bwt_mask[9][0], 0XE000);
    EXPECT_EQ(bwt_mask[9][1], 0XC000);
    EXPECT_EQ(bwt_mask[9][2], 0XC000);
    EXPECT_EQ(bwt_mask[9][3], 0XC000);

    EXPECT_EQ(bwt_mask[10][0], 0XE000);
    EXPECT_EQ(bwt_mask[10][1], 0XE000);
    EXPECT_EQ(bwt_mask[10][2], 0XC000);
    EXPECT_EQ(bwt_mask[10][3], 0XC000);

    EXPECT_EQ(bwt_mask[11][0], 0XE000);
    EXPECT_EQ(bwt_mask[11][1], 0XE000);
    EXPECT_EQ(bwt_mask[11][2], 0XE000);
    EXPECT_EQ(bwt_mask[11][3], 0XC000);

    EXPECT_EQ(bwt_mask[12][0], 0XE000);
    EXPECT_EQ(bwt_mask[12][1], 0XE000);
    EXPECT_EQ(bwt_mask[12][2], 0XE000);
    EXPECT_EQ(bwt_mask[12][3], 0XE000);

    EXPECT_EQ(bwt_mask[13][0], 0XF000);
    EXPECT_EQ(bwt_mask[13][1], 0XE000);
    EXPECT_EQ(bwt_mask[13][2], 0XE000);
    EXPECT_EQ(bwt_mask[13][3], 0XE000);

    EXPECT_EQ(bwt_mask[14][0], 0XF000);
    EXPECT_EQ(bwt_mask[14][1], 0XF000);
    EXPECT_EQ(bwt_mask[14][2], 0XE000);
    EXPECT_EQ(bwt_mask[14][3], 0XE000);

    EXPECT_EQ(bwt_mask[15][0], 0XF000);
    EXPECT_EQ(bwt_mask[15][1], 0XF000);
    EXPECT_EQ(bwt_mask[15][2], 0XF000);
    EXPECT_EQ(bwt_mask[15][3], 0XE000);

    EXPECT_EQ(bwt_mask[16][0], 0XF000);
    EXPECT_EQ(bwt_mask[16][1], 0XF000);
    EXPECT_EQ(bwt_mask[16][2], 0XF000);
    EXPECT_EQ(bwt_mask[16][3], 0XF000);

    EXPECT_EQ(bwt_mask[60][0], 0XFFFE);
    EXPECT_EQ(bwt_mask[60][1], 0XFFFE);
    EXPECT_EQ(bwt_mask[60][2], 0XFFFE);
    EXPECT_EQ(bwt_mask[60][3], 0XFFFE);

    EXPECT_EQ(bwt_mask[61][0], 0XFFFF);
    EXPECT_EQ(bwt_mask[61][1], 0XFFFE);
    EXPECT_EQ(bwt_mask[61][2], 0XFFFE);
    EXPECT_EQ(bwt_mask[61][3], 0XFFFE);

    EXPECT_EQ(bwt_mask[62][0], 0XFFFF);
    EXPECT_EQ(bwt_mask[62][1], 0XFFFF);
    EXPECT_EQ(bwt_mask[62][2], 0XFFFE);
    EXPECT_EQ(bwt_mask[62][3], 0XFFFE);

    EXPECT_EQ(bwt_mask[63][0], 0XFFFF);
    EXPECT_EQ(bwt_mask[63][1], 0XFFFF);
    EXPECT_EQ(bwt_mask[63][2], 0XFFFF);
    EXPECT_EQ(bwt_mask[63][3], 0XFFFE);
}
TEST_F(BackwardTest, testcase2)
{
    ;
}
