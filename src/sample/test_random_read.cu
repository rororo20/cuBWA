#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define DATA_SIZE         (1024 * 1024 * 512)  // 测试512MB数据
#define THREADS_PER_BLOCK 32
#define PER_READ_BYTES    64  // 每个wrap读的字节数量

// 生成随机访问模式的核函数
__global__ void generate_random_indices(unsigned int* indices, int size, unsigned long long seed)
{
    int tid = blockIdx.x;
    if (tid >= size) return;

    hiprandState_t state;
    hiprand_init(seed, tid, 0, &state);
    indices[tid] = (hiprand(&state) % size);
}

// 随机读写测试核函数
__global__ void random_access_kernel(char* data, unsigned int* indices, int size)
{
    int tid = blockIdx.x;
    if (tid >= size) return;
    int reg = data[indices[tid] * PER_READ_BYTES + threadIdx.x << 1];
    data[indices[tid] * PER_READ_BYTES + threadIdx.x << 1] = reg + 1;
}

int main()
{
    char* d_data;
    unsigned int* d_indices;
    hipEvent_t start, stop;
    float elapsed_time;

    // 分配设备内存
    hipMalloc(&d_data, DATA_SIZE * sizeof(char) * PER_READ_BYTES);
    hipMalloc(&d_indices, DATA_SIZE * sizeof(unsigned int));

    // 初始化数据
    hipMemset(d_data, 0, DATA_SIZE * sizeof(float));

    // 创建CUDA事件
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // 生成随机索引
    dim3 block(THREADS_PER_BLOCK);
    dim3 grid_generate(DATA_SIZE);
    generate_random_indices<<<grid_generate, block>>>(d_indices, DATA_SIZE, time(NULL));
    hipDeviceSynchronize();

    // 执行测试
    hipEventRecord(start);
    random_access_kernel<<<grid_generate, block>>>(d_data, d_indices, DATA_SIZE);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);

    // 计算带宽
    size_t bytes_accessed = DATA_SIZE * sizeof(char) * PER_READ_BYTES * 2;  // 读+写
    double bandwidth = (bytes_accessed / (elapsed_time / 1000.0)) / (1024.0 * 1024.0 * 1024.0);

    printf("Elapsed Time: %.3f ms\n", elapsed_time);
    printf("Effective Bandwidth: %.2f GB/s\n", bandwidth);

    // 清理资源
    hipFree(d_data);
    hipFree(d_indices);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
